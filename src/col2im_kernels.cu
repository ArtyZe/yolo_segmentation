#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "hipblas.h"

extern "C" {
#include "col2im.h"
#include "hip/hip_runtime.h"
}

// src: https://github.com/BVLC/caffe/blob/master/src/caffe/util/im2col.cu
// You may also want to read: https://github.com/BVLC/caffe/blob/master/LICENSE

__global__ void col2im_gpu_kernel(const int n, const float* data_col,
        const int height, const int width, const int ksize,
        const int pad,
        const int stride,
<<<<<<< HEAD
        const int height_col, const int width_col, const int dilation,
=======
        const int height_col, const int width_col,
>>>>>>> d2bad383be6fc51a225bdc438fe8661eec5816ee
        float *data_im) {
    int index = blockIdx.x*blockDim.x+threadIdx.x;
    for(; index < n; index += blockDim.x*gridDim.x){
        float val = 0;
        int w = index % width + pad;
        int h = (index / width) % height + pad;
        int c = index / (width * height);
        // compute the start and end of the output
<<<<<<< HEAD
        int w_col_start = (w < ksize) ? 0 : (w - ((ksize-1)*dilation+1)) / stride + 1;
        int w_col_end = min(w / stride + 1, width_col);
        int h_col_start = (h < ksize) ? 0 : (h - ((ksize-1)*dilation+1)) / stride + 1;
        int h_col_end = min(h / stride + 1, height_col);
        // equivalent implementation
        //int offset =
            //(c * ksize * ksize + h * ksize + w) * height_col * width_col;
        //int coeff_h_col = (1 - stride * ksize * height_col) * width_col;
        //int coeff_w_col = (1 - stride * height_col * width_col);
        //for (int h_col = h_col_start; h_col < h_col_end; ++h_col) {
            //for (int w_col = w_col_start; w_col < w_col_end; ++w_col) {
                //val += data_col[offset + h_col * coeff_h_col + w_col * coeff_w_col];
            //}
        //}
        //data_im[index] += val;
        for (int h_col = h_col_start; h_col < h_col_end; h_col += 1) {
		  for (int w_col = w_col_start; w_col < w_col_end; w_col += 1) {
			int h_k = (h - h_col * stride);
			int w_k = (w - w_col * stride);
			if (h_k % dilation == 0 && w_k % dilation == 0) {
			  h_k /= dilation;
			  w_k /= dilation;
			  int data_col_index = (((c * ksize + h_k) * ksize + w_k) *
									height_col + h_col) * width_col + w_col;
			  val += data_col[data_col_index];
			}
		  }
		}
		data_im[index] = val;
=======
        int w_col_start = (w < ksize) ? 0 : (w - ksize) / stride + 1;
        int w_col_end = min(w / stride + 1, width_col);
        int h_col_start = (h < ksize) ? 0 : (h - ksize) / stride + 1;
        int h_col_end = min(h / stride + 1, height_col);
        // equivalent implementation
        int offset =
            (c * ksize * ksize + h * ksize + w) * height_col * width_col;
        int coeff_h_col = (1 - stride * ksize * height_col) * width_col;
        int coeff_w_col = (1 - stride * height_col * width_col);
        for (int h_col = h_col_start; h_col < h_col_end; ++h_col) {
            for (int w_col = w_col_start; w_col < w_col_end; ++w_col) {
                val += data_col[offset + h_col * coeff_h_col + w_col * coeff_w_col];
            }
        }
        data_im[index] += val;
>>>>>>> d2bad383be6fc51a225bdc438fe8661eec5816ee
    }
}

void col2im_gpu(float *data_col,
        int channels, int height, int width,
<<<<<<< HEAD
        int ksize, int stride, int pad, int dilation, float *data_im){
    // We are going to launch channels * height_col * width_col kernels, each
    // kernel responsible for copying a single-channel grid.
    int height_col = (height + 2 * pad - (dilation * (ksize - 1) + 1)) / stride + 1;
    int width_col = (width + 2 * pad - (dilation * (ksize - 1) + 1)) / stride + 1;
=======
        int ksize, int stride, int pad, float *data_im){
    // We are going to launch channels * height_col * width_col kernels, each
    // kernel responsible for copying a single-channel grid.
    int height_col = (height + 2 * pad - ksize) / stride + 1;
    int width_col = (width + 2 * pad - ksize) / stride + 1;
>>>>>>> d2bad383be6fc51a225bdc438fe8661eec5816ee
    int num_kernels = channels * height * width;
    col2im_gpu_kernel<<<(num_kernels+BLOCK-1)/BLOCK,
        BLOCK>>>(
                num_kernels, data_col, height, width, ksize, pad,
                stride, height_col,
<<<<<<< HEAD
                width_col, dilation, data_im);
=======
                width_col, data_im);
>>>>>>> d2bad383be6fc51a225bdc438fe8661eec5816ee
}

