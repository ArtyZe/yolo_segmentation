#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "hipblas.h"

extern "C" {
#include "im2col.h"
#include "hip/hip_runtime.h"
}

// src: https://github.com/BVLC/caffe/blob/master/src/caffe/util/im2col.cu
// You may also want to read: https://github.com/BVLC/caffe/blob/master/LICENSE

__global__ void im2col_gpu_kernel(const int n, const float* data_im,
        const int height, const int width, const int ksize,
        const int pad,
        const int stride,
<<<<<<< HEAD
        const int height_col, const int width_col, const int dilation,
=======
        const int height_col, const int width_col,
>>>>>>> d2bad383be6fc51a225bdc438fe8661eec5816ee
        float *data_col) {
    int index = blockIdx.x*blockDim.x+threadIdx.x;
    for(; index < n; index += blockDim.x*gridDim.x){
        int w_out = index % width_col;
        int h_index = index / width_col;
        int h_out = h_index % height_col;
        int channel_in = h_index / height_col;
        int channel_out = channel_in * ksize * ksize;
        int h_in = h_out * stride - pad;
        int w_in = w_out * stride - pad;
        float* data_col_ptr = data_col;
        data_col_ptr += (channel_out * height_col + h_out) * width_col + w_out;
        const float* data_im_ptr = data_im;
        data_im_ptr += (channel_in * height + h_in) * width + w_in;
        for (int i = 0; i < ksize; ++i) {
            for (int j = 0; j < ksize; ++j) {
<<<<<<< HEAD
                int h = h_in + i * dilation;
                int w = w_in + j * dilation;

                *data_col_ptr = (h >= 0 && w >= 0 && h < height && w < width) ?
                    data_im_ptr[i * dilation * width + j * dilation] : 0;
=======
                int h = h_in + i;
                int w = w_in + j;

                *data_col_ptr = (h >= 0 && w >= 0 && h < height && w < width) ?
                    data_im_ptr[i * width + j] : 0;
>>>>>>> d2bad383be6fc51a225bdc438fe8661eec5816ee

                //*data_col_ptr = data_im_ptr[ii * width + jj];

                data_col_ptr += height_col * width_col;
            }
        }
    }
}

void im2col_gpu(float *im,
         int channels, int height, int width,
<<<<<<< HEAD
         int ksize, int stride, int pad, int dilation, float *data_col){
    // We are going to launch channels * height_col * width_col kernels, each
    // kernel responsible for copying a single-channel grid.
    int height_col = (height + 2 * pad - (dilation * (ksize - 1) + 1)) / stride + 1;
    int width_col = (width + 2 * pad - (dilation * (ksize - 1) + 1)) / stride + 1;
=======
         int ksize, int stride, int pad, float *data_col){
    // We are going to launch channels * height_col * width_col kernels, each
    // kernel responsible for copying a single-channel grid.
    int height_col = (height + 2 * pad - ksize) / stride + 1;
    int width_col = (width + 2 * pad - ksize) / stride + 1;
>>>>>>> d2bad383be6fc51a225bdc438fe8661eec5816ee
    int num_kernels = channels * height_col * width_col;
    im2col_gpu_kernel<<<(num_kernels+BLOCK-1)/BLOCK,
        BLOCK>>>(
                num_kernels, im, height, width, ksize, pad,
                stride, height_col,
<<<<<<< HEAD
                width_col, dilation, data_col);
=======
                width_col, data_col);
>>>>>>> d2bad383be6fc51a225bdc438fe8661eec5816ee
}
